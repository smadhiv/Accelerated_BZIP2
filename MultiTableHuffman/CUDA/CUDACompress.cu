#include "hip/hip_runtime.h"
/*---------------------------------------------------------------------------------------------------------------------------------------------*/
//Sriram Madhivanan
//GPU Implementation
/*---------------------------------------------------------------------------------------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <limits.h>
#include "../include/parallelHeader.h"
#define block_size 1024
#define MIN_SCRATCH_SIZE 50 * 1024 * 1024

struct huffmanTree *head_huffmanTreeNode;
struct huffmanTree huffmanTreeNode[512];
unsigned char bitSequenceConstMemory[256][255];
struct huffmanDictionary huffmanDictionary;
unsigned int constMemoryFlag = 0;

int main(int argc, char **argv){
	unsigned int i;
	unsigned int distinctCharacterCount, combinedHuffmanNodes, inputFileLength, frequency[256];
	unsigned char *inputFileData, bitSequenceLength = 0, bitSequence[255];
	unsigned int *compressedDataOffset, cpu_time_used;
	unsigned int integerOverflowFlag;
	FILE *inputFile, *compressedFile;
	long unsigned int mem_free, mem_total;
	long unsigned int mem_req, mem_offset, mem_data;
	int numKernelRuns;
	clock_t start, end;
	
	// check number of args
	if(argc != 3){
		printf("try with arguments InputFile and OutputFile");
		return -1;
	}
	// read input file, get inputFileLength and data
	inputFile = fopen(argv[1], "rb");
	fseek(inputFile, 0, SEEK_END);
	inputFileLength = ftell(inputFile);
	fseek(inputFile, 0, SEEK_SET);
	inputFileData = (unsigned char *)malloc(inputFileLength * sizeof(unsigned char));
	fread(inputFileData, sizeof(unsigned char), inputFileLength, inputFile);
	fclose(inputFile);
	
	// calculate run duration
	start = clock();
	
	// find the frequency of each symbols
	for (i = 0; i < 256; i++){
		frequency[i] = 0;
	}
	for (i = 0; i < inputFileLength; i++){
		frequency[inputFileData[i]]++;
	}

	// initialize nodes of huffman tree
	distinctCharacterCount = 0;
	for (i = 0; i < 256; i++){
		if (frequency[i] > 0){
			huffmanTreeNode[distinctCharacterCount].count = frequency[i];
			huffmanTreeNode[distinctCharacterCount].letter = i;
			huffmanTreeNode[distinctCharacterCount].left = NULL;
			huffmanTreeNode[distinctCharacterCount].right = NULL;
			distinctCharacterCount++;
		}
	}
	
	// build tree 
	for (i = 0; i < distinctCharacterCount - 1; i++){
		combinedHuffmanNodes = 2 * i;
		sortHuffmanTree(i, distinctCharacterCount, combinedHuffmanNodes);
		buildHuffmanTree(i, distinctCharacterCount, combinedHuffmanNodes);
	}
	
	// build table having the bitSequence sequence and its length
	buildHuffmanDictionary(head_huffmanTreeNode, bitSequence, bitSequenceLength);
	
	// calculate memory requirements
	// GPU memory
	hipMemGetInfo(&mem_free, &mem_total);
	
	// debug
	if(1){
		printf("Free Mem: %lu\n", mem_free);		
	}

	// offset array requirements
	mem_offset = 0;
	for(i = 0; i < 256; i++){
		mem_offset += frequency[i] * huffmanDictionary.bitSequenceLength[i];
	}
	mem_offset = mem_offset % 8 == 0 ? mem_offset : mem_offset + 8 - mem_offset % 8;
	
	// other memory requirements
	mem_data = inputFileLength + (inputFileLength + 1) * sizeof(unsigned int) + sizeof(huffmanDictionary);
	
	if(mem_free - mem_data < MIN_SCRATCH_SIZE){
		printf("\nExiting : Not enough memory on GPU\nmem_free = %lu\nmin_mem_req = %lu\n", mem_free, mem_data + MIN_SCRATCH_SIZE);
		return -1;
	}
	mem_req = mem_free - mem_data - 10 * 1024 * 1024;
	numKernelRuns = ceil((double)mem_offset / mem_req);
	integerOverflowFlag = mem_req + 255 <= UINT_MAX || mem_offset + 255 <= UINT_MAX ? 0 : 1;

	// debug
	if(1){
	printf("	InputFileSize      =%u\n\
	OutputSize         =%u\n\
	NumberOfKernel     =%d\n\
	integerOverflowFlag=%d\n", inputFileLength, mem_offset/8, numKernelRuns, integerOverflowFlag);		
	}

	
	// generate data offset array
	compressedDataOffset = (unsigned int *)malloc((inputFileLength + 1) * sizeof(unsigned int));

	// launch kernel
	lauchCUDAHuffmanCompress(inputFileData, compressedDataOffset, inputFileLength, numKernelRuns, integerOverflowFlag, mem_req);

	// calculate run duration
	end = clock();
	
	// write src inputFileLength, header and compressed data to output file
	compressedFile = fopen(argv[2], "wb");
	fwrite(&inputFileLength, sizeof(unsigned int), 1, compressedFile);
	fwrite(frequency, sizeof(unsigned int), 256, compressedFile);
	fwrite(inputFileData, sizeof(unsigned char), mem_offset / 8, compressedFile);
	fclose(compressedFile);	
	
	cpu_time_used = ((end - start)) * 1000 / CLOCKS_PER_SEC;
	printf("Time taken: %d:%d s\n", cpu_time_used / 1000, cpu_time_used % 1000);
	free(inputFileData);
	free(compressedDataOffset);
	return 0;
}
